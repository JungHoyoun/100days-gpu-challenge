#include <hip/hip_runtime.h>
#include <cstdio>
#include <vector>
#include <cstdlib>

#define CUDA_CHECK(stmt) do {                                   \
  hipError_t err = (stmt);                                     \
  if (err != hipSuccess) {                                     \
    std::fprintf(stderr, "CUDA error: %s (%s:%d)\n",            \
      hipGetErrorString(err), __FILE__, __LINE__);             \
    std::exit(EXIT_FAILURE);                                    \
  }                                                             \
} while (0)

__global__ void add1(int* d, int n){
  int i = blockIdx.x * blockDim.x + threadIdx.x;
  if (i < n) d[i] += 1;
}

int main(){
  const int n = 1 << 20;
  std::vector<int> h(n, 41);

  int* d = nullptr;
  CUDA_CHECK(hipMalloc(&d, n * sizeof(int)));
  CUDA_CHECK(hipMemcpy(d, h.data(), n * sizeof(int), hipMemcpyHostToDevice));

  int threads = 256;
  int blocks  = (n + threads - 1) / threads;
  add1<<<blocks, threads>>>(d, n);
  CUDA_CHECK(hipGetLastError());
  CUDA_CHECK(hipDeviceSynchronize());

  CUDA_CHECK(hipMemcpy(h.data(), d, n * sizeof(int), hipMemcpyDeviceToHost));
  CUDA_CHECK(hipFree(d));

  // 빠른 검증
  for(int i=0;i<10;++i) if(h[i] != 42){ std::fprintf(stderr,"mismatch\n"); return 1; }
  std::printf("ok\n");
  return 0;
}
